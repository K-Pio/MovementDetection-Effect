#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/opencv.hpp>

__global__ void substractKernel( unsigned char* f1, unsigned char* f2, unsigned char* nf, int width, int height );

hipError_t substractWithCuda( int width, int height, cv::Mat frame1, cv::Mat frame2, cv::Mat* matrix );